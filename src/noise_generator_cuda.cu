#include "hip/hip_runtime.h"
#include "noise_generator_cuda.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <vector>
#include <cstdio>
#include <cstdlib>
#include <chrono>

// Define constants
#define BLOCK_SIZE 256  // Number of threads per block

// CUDA error checking macro
#define CUDA_CALL(func)                                                         \
    {                                                                           \
        hipError_t err = (func);                                               \
        if (err != hipSuccess) {                                               \
            printf("CUDA error in file '%s' at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                                                 \
        }                                                                       \
    }

// Kernel to generate random noise
__global__ void generate_noise(double* noise_array, int num_samples, double mean, double stddev, unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_samples) return;

    // Initialize CURAND state
    hiprandState state;
    hiprand_init(seed + idx, 0, 0, &state);

    // Generate Gaussian noise
    noise_array[idx] = mean + stddev * hiprand_normal_double(&state);
}

void generateNoiseCUDA(std::vector<double>& noise, double mean, double stddev) {
    int num_samples = noise.size();
    double* d_noise_array;

    // Allocate memory on the device
    CUDA_CALL(hipMalloc((void**)&d_noise_array, num_samples * sizeof(double)));

    // Get CPU time as seed (using std::chrono)
    unsigned long long seed = std::chrono::system_clock::now().time_since_epoch().count();

    // Launch kernel with enough blocks to cover all samples
    int num_blocks = (num_samples + BLOCK_SIZE - 1) / BLOCK_SIZE;
    generate_noise<<<num_blocks, BLOCK_SIZE>>>(d_noise_array, num_samples, mean, stddev, seed);
    CUDA_CALL(hipGetLastError());
    CUDA_CALL(hipDeviceSynchronize());

    // Copy data back to the host
    CUDA_CALL(hipMemcpy(noise.data(), d_noise_array, num_samples * sizeof(double), hipMemcpyDeviceToHost));

    // Free device memory
    CUDA_CALL(hipFree(d_noise_array));
}
